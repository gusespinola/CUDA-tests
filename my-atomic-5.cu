#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1<<7
#define THREADS_PER_BLOCK 1024

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= N) return;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();
    if (0 == threadIdx.x) {
        float sum = 0.0;
        int max = THREADS_PER_BLOCK;
        if (N < max) max = N;

        for (int i = 0; i < max; i++) {
            sum += temp[i];
        }
        //c[0] = sum;
        atomicAdd(c, sum);
    }
}

void random_floats(float *a, float size)
{
    int i;
    for (i=0; i<size; i++)
        a[i] = i;
    return;
}

int main(void) {
    int i;
    float result;
    float *a, *b, *c; // host copies of a, b, c
    float *dev_a, *dev_b, *dev_c; // device copies of a, b, c
    int size = N * sizeof(float); // we need space for N floats
    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, sizeof(float) );
    a = (float*)malloc( size );
    b = (float*)malloc( size );
    c = (float*)malloc( sizeof(float) );

    random_floats( a, N );
    random_floats( b, N );
    /*
    printf("a = ");
    for (i=0; i<N; i++) printf("%d, ", a[i]);
    printf("\n");
    printf("b = ");
    for (i=0; i<N; i++) printf("%d, ", b[i]);
    printf("\n");
    */
    result = 0;
    for (i=0; i<N; i++) result += a[i] * b[i];
    *c = 0;

    // copy inputs to device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice);

    int blocks = (int)(N/THREADS_PER_BLOCK) + 1; // ceil(...)
    //if(blocks<1) blocks=1;

    // launch dot() kernel
    dot <<< blocks, THREADS_PER_BLOCK >>> (dev_a, dev_b, dev_c);

    // copy device result back to host copy of c
    hipMemcpy(c, dev_c, sizeof(float) , hipMemcpyDeviceToHost);

    printf("*c     = %.2f\n", *c);
    printf("result = %.2f\n", result);

    free(a); free(b); free(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

